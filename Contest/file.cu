#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>

#define N 1024
#define threadsPerBlock 512

__global__ void gpu_dot(float *d_a, float *d_b, float *d_c) 
{
  //Define Shared Memory
  __shared__ float partial_sum[threadsPerBlock];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int index = threadIdx.x;
  float sum = 0;
  while (tid < N) 
  {
    sum += d_a[tid] * d_b[tid];
    tid += blockDim.x * gridDim.x;
  }
  // set the partial sum in shared memory
  partial_sum[index] = sum;
  // synchronize threads in this block
  __syncthreads();
  //Calculate Patial sum for a current block using data in shared memory
        int i = blockDim.x / 2;
  while (i != 0) {
    if (index < i)
      {partial_sum[index] += partial_sum[index + i];}
    __syncthreads();
    i /= 2;
  }
  //Store result of partial sum for a block in global memory
  if (index == 0)
    d_c[blockIdx.x] = partial_sum[0];

}
