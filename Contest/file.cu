#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void arrmul(int *x,int *y, int *z)    //kernel definition
{
  int id=blockIdx.x; 
/* blockIdx.x gives the respective block id which starts from 0 */
  z[id]=x[id]*y[id];
}

int main()
{
    int a[6]={10,20,30,40,50,60};
    int b[6]={1,2,3,4,5,6};
    int c[6];
    int *d,*e,*f;
    int i;
 
/* cudaMalloc() allocates memory from Global memory on GPU */ 
    hipMalloc((void **)&d,6*sizeof(int)); 
    hipMalloc((void **)&e,6*sizeof(int));
    hipMalloc((void **)&f,6*sizeof(int));

/* cudaMemcpy() copies the contents from destination to source. Here destination is GPU(d,e) and source is CPU(a,b) */ 
 hipMemcpy(d,a,6*sizeof(int),hipMemcpyHostToDevice);    
 hipMemcpy(e,b,6*sizeof(int),hipMemcpyHostToDevice); 
 
/* call to kernel. Here 6 is number of blocks, 1 is the number of threads per block and d,e,f are the arguments */  
arrmul<<<6,1>>>(d,e,f); 

/* Here we are copying content from GPU(Device) to CPU(Host) */
 hipMemcpy(c,f,6*sizeof(int),hipMemcpyDeviceToHost); 
    
printf("\Dot Product of two arrays:\n ");
    for(i=0;i<6;i++)
    {
        printf("%d\t",c[i]);
    }

/* Free the memory allocated to pointers d,e,f */ 
    hipFree(d); 
    hipFree(e);
    hipFree(f);
    return 0;
}
